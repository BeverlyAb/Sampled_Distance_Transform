#include "hip/hip_runtime.h"
/*
Copyright (C) 2006 Pedro Felzenszwalb

This program is free software; you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation; either version 2 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307 USA
*/

#include <cstdio>
#include <cstdlib>
#include <cmath>
#include "pnmfile.h"
#include "imconv.h"
#include "dt.h"
#include "cuda_utils.h"
typedef uchar dtype;
#define MAX_THREADS 256


//kernel0 (dtype *input, dtype *output, unsigned int n)
__global__ void
kernel_threshold(uchar *im_ptr,uchar *out_ptr, int height,int width)
{
  //Idea is for each thread to do the dt computation save back in scratch and then 

  __shared__  dtype scratch[height * width];

  unsigned int bid = gridDim.x * blockIdx.y + blockIdx.x;
  unsigned int i = bid * blockDim.x + threadIdx.x;

  if(i<height)
  {
    for(int j=0;j<width;j++)
    {
      if(j<200)
      {
        scratch[i*width +j] = 90;
      }
      else
      {
        scratch[i*width +j] = *(im_ptr + i*width+j);
      }
     // scratch[i*width +j] = *(im_ptr + i*width+j);
      
    }
  }


  __syncthreads ();

  // for(int i=0;i<244;i++)
  // {
  //   for(int j=0;j<200;j++)
  //   {
  //     *(im_ptr + 400*i + j) = 90;
  //   }
  // }

  
  if(i<height)
  {
    for(int j=0;j<width;j++)
    {
      *(out_ptr + i*width +j) = scratch[i*width +j]; 
    }
  }


  printf("value= %f", im_ptr[0]);
}



int main(int argc, char **argv) {
  if (argc != 3) {
    fprintf(stderr, "usage: %s input(pbm) output(pgm)\n", argv[0]);
    return 1;
  }

  char *input_name = argv[1];
  char *output_name = argv[2];

  dtype *h_idata, *h_odata, h_cpu;
  dtype *d_idata, *d_odata;

  float *temp_arr;
  temp_arr = new float[1];
  temp_arr[0] = 5;
  //threshold(temp_arr);
  // load input
  image<uchar> *input = loadPGM(input_name);

  // compute dt
  //image<float> *out = dt(input);

  threshold(input->data);
  int N = input->height() * input->width();


  int blocks = ceil(input->height()/MAX_THREADS);
  h_idata = (dtype*) malloc (N * sizeof (dtype));
  CUDA_CHECK_ERROR (hipMalloc (&d_idata, N * sizeof (dtype)));
  CUDA_CHECK_ERROR (hipMalloc (&d_odata, N * sizeof (dtype)));

  // /* Initialize array */
  // srand48(time(NULL));
  // for(i = 0; i < N; i++) {
  //   h_idata[i] = drand48() / 100000;
  // }
  CUDA_CHECK_ERROR (hipMemcpy (d_idata, input->data, N * sizeof (dtype), 
				hipMemcpyHostToDevice));


  kernel_threshold <<<gb, tb>>> (d_idata, d_odata, input->height(),input->width());



  CUDA_CHECK_ERROR (hipMemcpy (h_odata, d_odata, sizeof (dtype), 
  hipMemcpyDeviceToHost));
  // // take square roots
  // for (int y = 0; y < input->height(); y++) {
  //   for (int x = 0; x < out->width(); x++) {
  //     imRef(input, x, y) = sqrt(imRef(out, x, y));
  //   }
  // }

  // convert to grayscale
//  image<uchar> *gray = imageFLOATtoUCHAR(out);
  input->data = h_odata;
  // save output
  savePGM(input, output_name);

  delete input;
 // delete out;
 // delete gray;
}
